/*
 *     
 *  IMAGE PROCESSING
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define pixel(i, j, n)  (((j)*(n)) +(i))
int B = 16;


/*read*/
void  readimg(char * filename,int nx, int ny, int * image){
  
   FILE *fp=NULL;

   fp = fopen(filename,"r");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fscanf(fp,"%d", &image[pixel(i,j,nx)]);      
      }
   }
   fclose(fp);
}

/* save */   
void saveimg(char *filename,int nx,int ny,int *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fprintf(fp,"%d ", image[pixel(i,j,nx)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);

}

/*invert*/
__global__ void invert(int* image, int* image_invert, int nx, int ny){
  int indx = threadIdx.x + blockIdx.x * blockDim.x;
  int indy = threadIdx.y + blockIdx.y * blockDim.y;
  if(indx>=0 && indx < nx){
    if(indy>=0 && indy < ny){
      image_invert[pixel(indx,indy,nx)] = 255 - image[pixel(indx,indy,nx)];
    }
  }
}

/*smooth*/
__global__ void smooth(int* image, int* image_smooth, int nx, int ny){
  int indx = threadIdx.x + blockIdx.x * blockDim.x;
  int indy = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(indx >= 0 && indx < nx){
    if(indy >= 0 && indy < ny){
      if(indx == 0 || indy == 0 || indx == nx-1 || indy == ny-1){
        image_smooth[pixel(indx,indy,nx)] = 0;
      }
      else{
        image_smooth[pixel(indx,indy,nx)] =(image[pixel(indx-1,indy+1,nx)] + image[pixel(indx,indy+1,nx)] +image[pixel(indx+1,indy+1,nx)] + image[pixel(indx-1,indy,nx)] + image[pixel(indx,indy,nx)] + image[pixel(indx+1,indy,nx)] + image[pixel(indx-1,indy-1,nx)] + image[pixel(indx,indy-1,nx)] + image[pixel(indx+1,indy-1,nx)]);
	image_smooth[pixel(indx,indy,nx)] = (int) image_smooth[pixel(indx,indy,nx)] / 9;
        if(image_smooth[pixel(indx, indy, nx)] < 0){
          image_smooth[pixel(indx, indy, nx)] = 0;
        } else if (image_smooth[pixel(indx, indy, nx)] > 255){
          image_smooth[pixel(indx, indy, nx)] = 255;
        }
      }
    }
  }
}

/*detect*/
__global__ void detect(int* image, int* image_detect, int nx, int ny){
  int indx = threadIdx.x + blockIdx.x * blockDim.x;
  int indy = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(indx>=0 && indx < nx){
    if(indy >= 0 && indy < ny){
      if(indx == 0 || indy == 0 || indx == nx-1 || indy == ny-1){
        image_detect[pixel(indx,indy,nx)] = 0;
      }
      else{
        image_detect[pixel(indx,indy,nx)] = image[pixel(indx-1,indy,nx)] + image[pixel(indx+1,indy,nx)] + image[pixel(indx,indy-1,nx)] + image[pixel(indx,indy+1,nx)] - 4 * image[pixel(indx,indy,nx)];
        if(image_detect[pixel(indx, indy, nx)] < 0){
          image_detect[pixel(indx, indy, nx)] = 0;
        } else if (image_detect[pixel(indx, indy, nx)] > 255){
          image_detect[pixel(indx, indy, nx)] = 255;
        }
      }
    }
  }
}

/*enhance*/
__global__ void enhance(int* image,int *image_enhance,int nx, int ny){
  int indx = threadIdx.x + blockIdx.x * blockDim.x;
  int indy = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(indx >= 0 && indx < nx){
    if(indy >= 0 && indy < ny){
      if(indx == 0 || indy == 0 || indx == nx-1 || indy == ny-1){
        image_enhance[pixel(indx,indy,nx)] = 0;
      }
      else{
        image_enhance[pixel(indx,indy,nx)] = 5*image[pixel(indx,indy,nx)] -(image[pixel(indx-1,indy,nx)] +image[pixel(indx+1,indy,nx)] + image[pixel(indx,indy-1,nx)] + image[pixel(indx,indy+1,nx)]);
        if(image_enhance[pixel(indx, indy, nx)] < 0){
          image_enhance[pixel(indx, indy, nx)] = 0;
        } else if (image_enhance[pixel(indx, indy, nx)] > 255){
          image_enhance[pixel(indx, indy, nx)] = 255;
        }
      }
    }
  }   
}

/* Main program */
int main (int argc, char *argv[])
{
   int    nx,ny;
   char   filename[250];
   float runtime;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   /* Get parameters */
   if (argc != 4) 
   {
      printf ("Usage: %s image_name N M \n", argv[0]);
      exit (1);
   }
   sprintf(filename, "%s.txt", argv[1]);
   nx  = atoi(argv[2]);
   ny  = atoi(argv[3]);

   printf("%s %d %d\n", filename, nx, ny);

   /* Allocate CPU and GPU pointers */

   int*   image=(int *) malloc(sizeof(int)*nx*ny); 
   int*   image_invert  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_smooth  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_detect  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_enhance = (int *) malloc(sizeof(int)*nx*ny); 

   int*   d_image, *d_image_invert, *d_image_smooth, *d_image_detect, *d_image_enhance;
   hipMalloc((void **) &d_image, sizeof(int)*nx*ny); 
   hipMalloc((void **) &d_image_invert, sizeof(int)*nx*ny); 
   hipMalloc((void **) &d_image_smooth, sizeof(int)*nx*ny); 
   hipMalloc((void **) &d_image_detect, sizeof(int)*nx*ny); 
   hipMalloc((void **) &d_image_enhance, sizeof(int)*nx*ny);
  

   /* Read image and save in array imgage */
   readimg(filename,nx,ny,image);

   dim3 dimBlock(B,B,1);
   int dimgx = (nx+B-1)/B;
   int dimgy = (ny+B-1)/B;
   dim3 dimGrid(dimgx,dimgy, 1);

   hipEventRecord(start);

   hipMemcpy(d_image, image, sizeof(int)*nx*ny, hipMemcpyHostToDevice);
   invert<<<dimGrid, dimBlock>>>(d_image, d_image_invert, nx, ny);
   smooth<<<dimGrid, dimBlock>>>(d_image, d_image_smooth, nx, ny);
   detect<<<dimGrid, dimBlock>>>(d_image, d_image_detect, nx, ny);
   enhance<<<dimGrid, dimBlock>>>(d_image, d_image_enhance, nx, ny);
   hipMemcpy(image_invert, d_image_invert, sizeof(int)*nx*ny, hipMemcpyDeviceToHost);
   hipMemcpy(image_smooth, d_image_smooth, sizeof(int)*nx*ny, hipMemcpyDeviceToHost);
   hipMemcpy(image_detect, d_image_detect, sizeof(int)*nx*ny, hipMemcpyDeviceToHost);
   hipMemcpy(image_enhance, d_image_enhance, sizeof(int)*nx*ny, hipMemcpyDeviceToHost);

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&runtime, start, stop);

   printf("Total time: %f\n", runtime);

   /* Save images */
   char fileout[255]={0};
   sprintf(fileout, "%s-inverse.txt", argv[1]);
   saveimg(fileout,nx,ny,image_invert);
   sprintf(fileout, "%s-smooth.txt", argv[1]);
   saveimg(fileout,nx,ny,image_smooth);
   sprintf(fileout, "%s-detect.txt", argv[1]);
   saveimg(fileout,nx,ny,image_detect);
   sprintf(fileout, "%s-enhance.txt", argv[1]);
   saveimg(fileout,nx,ny,image_enhance);

   /* Deallocate CPU and GPU pointers*/
   free(image);
   free(image_invert);
   free(image_smooth);
   free(image_detect);
   free(image_enhance);

   hipFree(d_image);
   hipFree(d_image_invert);
   hipFree(d_image_smooth);
   hipFree(d_image_detect);
   hipFree(d_image_enhance);
}
